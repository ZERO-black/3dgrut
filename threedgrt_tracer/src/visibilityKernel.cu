#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <3dgrt/mathUtils.h>
#include <ATen/cuda/HIPContext.h>  // for at::cuda::getCurrentCUDAStream()
#include <3dgrt/visibilityKernel.h> // declare computeVisibilityKernel()
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_math_constants.h>

__global__ void computeVisibilityKernel(
    const float* lods,
    const float* extra_levels,
    const float3* gPos,
    unsigned char* mask,
    int count,
    float3 eye,
    float std_dist) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count)
        return;

    float3 anchor    = gPos[idx];
    float dist       = length(anchor - eye);
    float pred_level = log2f(std_dist / dist); //+ extra_levels[idx];
    mask[idx]        = (lods[idx] <= pred_level) ? 1 : 0;
}

inline uint32_t div_round_up(uint32_t x, uint32_t y) {
    return (x + y - 1) / y;
}

void launchVisibilityKernel(
    const float* lods,
    const float* extra_levels,
    const float3* gPos,
    unsigned char* mask,
    int count,
    float3 eye,
    float std_dist) {
    // 1. CUDA stream
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    // 2. Grid/block dimensions
    constexpr uint32_t threads = 1024;
    uint32_t blocks            = div_round_up(static_cast<uint32_t>(count), threads);

    // 3. Launch the visibility kernel
    computeVisibilityKernel<<<blocks, threads, 0, stream>>>(
        lods,
        extra_levels,
        gPos,
        mask,
        count,
        eye,
        std_dist);

    // 4. Error check and synchronize
    hipError_t err = hipPeekAtLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    hipStreamSynchronize(stream);
}
